#include "hip/hip_runtime.h"
#include "graph.h"
#include "sample.hh"
#include <hipcub/hipcub.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLK_SZ BLOCK_SIZE
typedef double CounterType;
typedef hipcub::BlockReduce<CounterType, BLK_SZ> BlockReduce;
void sample_clique(Graph &g, int k, eidType num_samples, uint64_t &estimate);
void sample_4clique(Graph &g, eidType num_samples, uint64_t &counter);

int main(int argc, char* argv[]) {
  if (argc < 4) {
    std::cout << "Usage: " << argv[0] << " <graph> <k> <num_samples>\n";
    std::cout << "Example: " << argv[0] << " ../../inputs/mico/graph 3 1000\n";
    exit(1);
  }
  Graph g(argv[1], USE_DAG);
  g.print_meta_data();

  int k = atoi(argv[2]);
  assert(k > 2);
  int64_t num_samples = atoi(argv[3]);

  g.init_simple_edgelist();
  std::cout << "Taking " << num_samples << " samples\n";

  Timer t;
  t.Start();
  uint64_t total = 0;
  sample_clique(g, k, num_samples, total);
  t.Stop();
  std::cout << "Total runtime [data-copy + gpu-compute] = " << t.Seconds() << " sec\n";
  std::cout << "Estimated count " << FormatWithCommas(total) << "\n";
}

#include <hipcub/hipcub.hpp>
#include "graph_gpu.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

#define window_size 16384

template <typename T>
__device__ T random_select_single(T n, hiprandState &state) {
  return T(ceil(hiprand_uniform(&state) * n) - 1);
}

__global__ void setup_kernel(hiprandState *states) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hiprand_init(1234, idx, 0, &states[idx]);
}

// for k-clique with k > 4
__global__ void k_clique_sample(int k, hiprandState* states, eidType n_samples, float* edges_rng, GraphGPU g, vidType *vlists, vidType max_deg, CounterType *total, AccType *num_hits) {
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int thread_id   = blockIdx.x * blockDim.x + threadIdx.x;
  int warp_id     = thread_id   / WARP_SIZE;                // global warp index
  int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
  int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
  int num_warps   = (BLK_SZ / WARP_SIZE) * gridDim.x;       // total number of active warps
  CounterType counter = 0;

  // double buffer for each warp
  vidType *vlist0 = &vlists[int64_t(warp_id)*int64_t(max_deg)*2];
  vidType *vlist1 = &vlist0[max_deg];

  __shared__ vidType list_size[WARPS_PER_BLOCK];
  __shared__ vidType idx1[WARPS_PER_BLOCK];
  __shared__ CounterType scale[WARPS_PER_BLOCK];

  // each warp draws one sample
  for (eidType sample_id = warp_id; sample_id < n_samples; sample_id += num_warps) {
    if (thread_lane == 0) scale[warp_lane] = 1;
    __syncwarp();

    // sample an edge
    eidType eid = ceil(edges_rng[sample_id]*g.E()) - 1;
    auto v0 = g.get_src(eid);
    auto v1 = g.get_dst(eid);
    vidType v0_size = g.getOutDegree(v0);
    vidType v1_size = g.getOutDegree(v1);

    // intersection of v0 and v1's neighbors
    auto deg0 = intersect(g.N(v0), v0_size, g.N(v1), v1_size, vlist0);
    if (deg0 < k-2) continue;

    // sample a common neighbor of v0 and v1 as v2
    if (thread_lane == 0) idx1[warp_lane] = ceil(hiprand_uniform(&states[warp_id]) * deg0) - 1;
    __syncwarp();
    vidType v = vlist0[idx1[warp_lane]];
    vidType v_deg = g.getOutDegree(v);
    if (thread_lane == 0) scale[warp_lane] *= deg0;

    auto adj0 = vlist0;
    auto adj1 = vlist1;
    for (int j = 2; j < k-1; j++) {
      if (j == k - 2) {
        auto num = intersect_num(g.N(v), v_deg, adj0, deg0);
        auto c = warp_reduce<AccType>(num);
        deg0 = c;
      } else {
        auto c = intersect(g.N(v), v_deg, adj0, deg0, adj1);
        if (c < k-j-1) {
          if (thread_lane == 0) scale[warp_lane] = 0;
          __syncwarp();
          break;
        }
        if (thread_lane == 0) idx1[warp_lane] = ceil(hiprand_uniform(&states[warp_id]) * c) - 1;
        __syncwarp();
        v = adj1[idx1[warp_lane]];
        v_deg = g.getOutDegree(v);

        // swap the buffers
        auto temp = adj0;
        adj0 = adj1;
        adj1 = temp;
        deg0 = c;
      }
      if (thread_lane == 0) scale[warp_lane] *= deg0;
      __syncwarp();
    }
    if (thread_lane == 0) atomicAdd(num_hits, 1);
    if (thread_lane == 0) counter += scale[warp_lane];
  }
  auto block_num = BlockReduce(temp_storage).Sum(counter);
  if (threadIdx.x == 0) atomicAdd(total, block_num);
}

__global__ void clique4_sample(hiprandState* states, eidType n_samples, float* edges_rng, GraphGPU g, vidType *vlists, vidType max_deg, CounterType *total, AccType *num_hits) {
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int thread_id   = blockIdx.x * blockDim.x + threadIdx.x;
  int warp_id     = thread_id   / WARP_SIZE;                // global warp index
  int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
  int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
  int num_warps   = (BLK_SZ / WARP_SIZE) * gridDim.x;       // total number of active warps
  vidType *vlist  = &vlists[int64_t(warp_id)*int64_t(max_deg)];
  CounterType counter = 0;
  __shared__ vidType list_size[WARPS_PER_BLOCK];
  __shared__ vidType idx1[WARPS_PER_BLOCK];
  if (thread_lane == 0) idx1[warp_lane] = 0;

//  __shared__ hiprandState state[WARPS_PER_BLOCK];
//  if (thread_lane == 0) hiprand_init(1234, warp_id, 0, &state[warp_lane]);
  //hiprandState state;
  //hiprand_init(1234, thread_id, 0, &state);

  // each warp draws one sample
  for (eidType sample_id = warp_id; sample_id < n_samples; sample_id += num_warps) {
    // sample an edge
    eidType eid = ceil(edges_rng[sample_id]*g.E()) - 1;
    auto v0 = g.get_src(eid);
    auto v1 = g.get_dst(eid);
    vidType v0_size = g.getOutDegree(v0);
    vidType v1_size = g.getOutDegree(v1);

    // intersection of v0 and v1's neighbors
    auto d1 = intersect(g.N(v0), v0_size, g.N(v1), v1_size, vlist);
    if (d1 < 2) continue;

    // sample a common neighbor of v0 and v1's as v2
    //if (thread_lane == 0) idx1[warp_lane] = random_select_single<vidType>(d1, state[warp_lane]);
    if (thread_lane == 0) idx1[warp_lane] = ceil(hiprand_uniform(&states[warp_id]) * d1) - 1;
    //if (thread_lane == 0) idx1[warp_lane] = 0;
    __syncwarp();
    vidType v2 = vlist[idx1[warp_lane]];
    vidType v2_degree = g.getOutDegree(v2);

    // intersection of v0, v1 and v2's neighbors
    auto num = intersect_num(vlist, d1, g.N(v2), v2_degree);
    AccType scale = warp_reduce<vidType>(num);
    if (thread_lane == 0) atomicAdd(num_hits, 1);
    if (thread_lane == 0) counter += scale * AccType(d1);
  }
  auto block_num = BlockReduce(temp_storage).Sum(counter);
  if (threadIdx.x == 0) atomicAdd(total, block_num);
}

#define PAR_FACTOR 32
void sample_clique(Graph &g, int k, eidType num_samples, uint64_t &estimate) {
  vidType nv = g.num_vertices();
  eidType ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t memsize = print_device_info(0);
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";
  if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

  Timer t;
  t.Start();
  GraphGPU gg(g);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "time[copy graph to gpu] = " << t.Seconds() << " sec\n";

  t.Start();
  //gg.init_edgelist(g);
  gg.copy_edgelist_to_device(g.E(), g);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "time[copy edgelist to gpu] = " << t.Seconds() << " sec\n";

  size_t nwarps = WARPS_PER_BLOCK;
  size_t nthreads = BLK_SZ;
  size_t nblocks = (num_samples-1)/WARPS_PER_BLOCK+1;
  int max_blocks_per_SM = 0;
  if (k > 4) max_blocks_per_SM = maximum_residency(k_clique_sample, nthreads, 0);
  else max_blocks_per_SM = maximum_residency(clique4_sample, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  size_t max_resident_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  std::cout << "max_resident_blocks = " << max_resident_blocks << "\n";
  if (nblocks > max_resident_blocks*PAR_FACTOR) nblocks = max_resident_blocks*PAR_FACTOR;
  if (nblocks > 65536) nblocks = 65536;

  size_t per_block_vlist_size = nwarps * size_t(md) * sizeof(vidType);
  size_t nb = (memsize - mem_graph) / per_block_vlist_size;
  if (nb < nblocks) nblocks = nb;

  size_t list_size = nblocks * per_block_vlist_size;
  if (k > 4) list_size *= 2;
  std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
  vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
  CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  CounterType h_counter = 0, *d_counter;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_counter, sizeof(CounterType)));
  CUDA_SAFE_CALL(hipMemcpy(d_counter, &h_counter, sizeof(CounterType), hipMemcpyHostToDevice));
  AccType h_num_hits = 0, *d_num_hits;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_num_hits, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_num_hits, &h_num_hits, sizeof(AccType), hipMemcpyHostToDevice));

  // generate random numbers for edge sampling
  t.Start();
  float* d_rands;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_rands, num_samples * sizeof(float)));
  hiprandGenerator_t curand_gen; // used to generate random numbers on GPU
  auto status = hiprandCreateGenerator (&curand_gen ,HIPRAND_RNG_PSEUDO_DEFAULT );  
  if (status != HIPRAND_STATUS_SUCCESS) {
    printf("Error encountered in generating handle\n");  
  }
  // set seed
  status = hiprandSetPseudoRandomGeneratorSeed(curand_gen ,1234ULL);  
  if (status != HIPRAND_STATUS_SUCCESS) {
    printf("Error encountered in setting seed\n");
  }
  // Generate N floats on device between 0.0 and 1.0
  CURAND_CHECK(hiprandGenerateUniform(curand_gen, d_rands, num_samples));
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "RNG per sample runtime [gpu] = " << t.Seconds() << " sec\n";

  // allocate a RNG for each warp
  hiprandState *d_states;
  hipMalloc(&d_states, sizeof(hiprandState)*nblocks*nwarps);
  setup_kernel<<<nblocks, nwarps>>>(d_states);
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  std::cout << "CUDA sampling " << k << "-clique (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  t.Start();
  std::cout << "Launching kernel: num_samples=" << num_samples << "\n";
  if (k == 4)
    clique4_sample<<<nblocks, nthreads>>>(d_states, num_samples, d_rands, gg, frontier_list, md, d_counter, d_num_hits);
  else
    k_clique_sample<<<nblocks, nthreads>>>(k, d_states, num_samples, d_rands, gg, frontier_list, md, d_counter, d_num_hits);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "runtime [gpu] = " << t.Seconds() << " sec\n";

  CUDA_SAFE_CALL(hipMemcpy(&h_counter, d_counter, sizeof(CounterType), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(&h_num_hits, d_num_hits, sizeof(AccType), hipMemcpyDeviceToHost));

  std::cout << "counter = " << h_counter << " avg_count_per_sample = " << double(h_counter)/double(num_samples) << "\n";
  std::cout << "num_hits = " << h_num_hits << " hit_rate = " << double(h_num_hits)/double(num_samples) << "\n";

  estimate = uint64_t(h_counter * g.E() / num_samples);

  // clean up
  CUDA_SAFE_CALL(hipFree(d_counter));
  CURAND_CHECK(hiprandDestroyGenerator(curand_gen));
  CUDA_SAFE_CALL(hipFree(d_rands));
}

