#include "hip/hip_runtime.h"
#include "graph.h"
#include "sample.hh"

double sample_path(Graph &g, int k, eidType num_samples);

int main(int argc, char* argv[]) {
  if (argc < 4) {
    std::cout << "Usage: " << argv[0] << " <graph> <path_nodes> <num_samples>\n";
    std::cout << "Example: " << argv[0] << " ../../inputs/mico/graph 3 1000\n";
    exit(1);
  }
  Graph g(argv[1]);
  g.print_meta_data();

  int k = atoi(argv[2]); // number of vertices in the path
  assert(k > 2);
  std::cout << "Finding " << k << "-path\n";
  eidType num_samples = atoi(argv[3]);
  std::cout << "num_samples: " << num_samples << "\n";

  Timer t;
  g.init_simple_edgelist();
  t.Start();
  double estimate = sample_path(g, k, num_samples);
  t.Stop();
  std::cout << "runtime = " << t.Seconds() << " sec\n";
  //std::cout << "Estimated count " << FormatWithCommas(estimate) << "\n";
  std::cout << "Estimated count " << estimate << "\n";
}

#include <hipcub/hipcub.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "graph_gpu.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

#define BLK_SZ BLOCK_SIZE
#define MAX_PATTERN_SIZE 9
typedef double CounterType;
typedef hipcub::BlockReduce<CounterType, BLK_SZ> BlockReduce;

template <typename T>
__device__ T random_select_single(T n, hiprandState &state) {
  return T(ceil(hiprand_uniform(&state) * n) - 1);
}

__global__ void setup_kernel(hiprandState *states) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hiprand_init(1234, idx, 0, &states[idx]);
}

__global__ void k_path_sample(int k, hiprandState* states, eidType n_samples, float* edges_rng, GraphGPU g, vidType *vlists, vidType max_deg, CounterType *total, AccType *num_hits) {
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int thread_id   = blockIdx.x * blockDim.x + threadIdx.x;
  int warp_id     = thread_id   / WARP_SIZE;                // global warp index
  int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
  int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
  int num_warps   = (BLK_SZ / WARP_SIZE) * gridDim.x;       // total number of active warps
  CounterType counter = 0;
  //vidType *vlist0 = &vlists[int64_t(warp_id)*int64_t(max_deg)];

  __shared__ vidType list_size[WARPS_PER_BLOCK];
  __shared__ vidType idx[WARPS_PER_BLOCK];
  __shared__ CounterType scale[WARPS_PER_BLOCK];
  __shared__ vidType vs[WARPS_PER_BLOCK][MAX_PATTERN_SIZE-1];
  __shared__ vidType vs_size[WARPS_PER_BLOCK];

  // each warp draws one sample
  for (eidType sample_id = warp_id; sample_id < n_samples; sample_id += num_warps) {
    if (thread_lane == 0) scale[warp_lane] = 1;
    __syncwarp();

    // sample an edge
    eidType eid = ceil(edges_rng[sample_id]*g.E()) - 1;
    auto v0 = g.get_src(eid);
    auto v1 = g.get_dst(eid);
    vidType v0_size = g.getOutDegree(v0);
    vidType v1_size = g.getOutDegree(v1);

    // buffer existing nodes
    if (thread_lane == 0) {
      vs[warp_lane][0] = v0 < v1 ? v0 : v1;
      vs[warp_lane][1] = v0 < v1 ? v1 : v0;
      vs_size[warp_lane] = 2;
    }
    __syncwarp();

    vidType v = v1;
    vidType v_deg = g.getOutDegree(v);

    for (int j = 2; j < k; j++) {
      vidType c = 0;
      if (j == k - 1) {
        auto num = difference_num(g.N(v), v_deg, vs[warp_lane], vs_size[warp_lane], v0);
        c = warp_reduce<vidType>(num);
      } else {
        //c = difference_set(g.N(v), v_deg, vs[warp_lane], vs_size[warp_lane], vlist0);
        auto num = difference_num(g.N(v), v_deg, vs[warp_lane], vs_size[warp_lane]);
        c = warp_reduce<vidType>(num);
        if (c < 1) {
          if (thread_lane == 0) scale[warp_lane] = 0; __syncwarp();
          break;
        }
        if (thread_lane == 0) idx[warp_lane] = ceil(hiprand_uniform(&states[warp_id]) * c) - 1;
        __syncwarp();
        //v = vlist0[idx[warp_lane]];
        v = difference_sample(g.N(v), v_deg, vs[warp_lane], vs_size[warp_lane], idx[warp_lane]);
        v_deg = g.getOutDegree(v);
        if (thread_lane == 0) {
          int i = 0;
          for (; i < j; i++) {
            if (v < vs[warp_lane][i]) {
              for (int ii = j-1; ii >= i; ii --) {
                vs[warp_lane][ii+1] = vs[warp_lane][ii];
              }
              break;
            }
          }
          vs[warp_lane][i] = v;
          vs_size[warp_lane] ++;
        }
        //warp_sort(vs[warp_lane], vs_size[warp_lane]);
        __syncwarp();
      }
      if (c < 1) {
        if (thread_lane == 0) scale[warp_lane] = 0;
        __syncwarp();
        break;
      }
      if (thread_lane == 0) scale[warp_lane] *= c;
      __syncwarp();
    }
    if (thread_lane == 0) atomicAdd(num_hits, 1);
    if (thread_lane == 0) counter += scale[warp_lane];
  }
  auto block_num = BlockReduce(temp_storage).Sum(counter);
  if (threadIdx.x == 0) atomicAdd(total, block_num);
}

#define PAR_FACTOR 32
double sample_path(Graph &g, int k, eidType num_samples) {
  vidType nv = g.num_vertices();
  eidType ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t memsize = print_device_info(0);
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";
  if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

  Timer t;
  t.Start();
  GraphGPU gg(g);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "time[copy graph to gpu] = " << t.Seconds() << " sec\n";

  t.Start();
  gg.copy_edgelist_to_device(g.E(), g);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "time[copy edgelist to gpu] = " << t.Seconds() << " sec\n";

  size_t nwarps = WARPS_PER_BLOCK;
  size_t nthreads = BLK_SZ;
  size_t nblocks = (num_samples-1)/WARPS_PER_BLOCK+1;
  int max_blocks_per_SM = 0;
  max_blocks_per_SM = maximum_residency(k_path_sample, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  size_t max_resident_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  std::cout << "max_resident_blocks = " << max_resident_blocks << "\n";
  if (nblocks > max_resident_blocks*PAR_FACTOR) nblocks = max_resident_blocks*PAR_FACTOR;
  if (nblocks > 65536) nblocks = 65536;

  std::cout << "CUDA sampling " << k << "-path (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  vidType *frontier_list; // each warp has a vertex set; each set has size of max_degree
/*
  size_t per_block_vlist_size = nwarps * size_t(md) * sizeof(vidType);
  std::cout << "per-block frontier list size: " << per_block_vlist_size/(1024*1024) << " MB\n";
  //size_t nb = (memsize - mem_graph) / per_block_vlist_size;
  //if (nb < nblocks) nblocks = nb;
  size_t list_size = nblocks * per_block_vlist_size;
  std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
  CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));
  CUDA_SAFE_CALL(hipDeviceSynchronize());
*/
  CounterType h_counter = 0, *d_counter;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_counter, sizeof(CounterType)));
  CUDA_SAFE_CALL(hipMemcpy(d_counter, &h_counter, sizeof(CounterType), hipMemcpyHostToDevice));
  AccType h_num_hits = 0, *d_num_hits;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_num_hits, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_num_hits, &h_num_hits, sizeof(AccType), hipMemcpyHostToDevice));

  // generate random numbers for edge sampling
  t.Start();
  float* d_rands;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_rands, num_samples * sizeof(float)));
  hiprandGenerator_t curand_gen; // used to generate random numbers on GPU
  auto status = hiprandCreateGenerator (&curand_gen ,HIPRAND_RNG_PSEUDO_DEFAULT );  
  if (status != HIPRAND_STATUS_SUCCESS) {
    printf("Error encountered in generating handle\n");  
  }
  // set seed
  status = hiprandSetPseudoRandomGeneratorSeed(curand_gen ,1234ULL);  
  if (status != HIPRAND_STATUS_SUCCESS) {
    printf("Error encountered in setting seed\n");
  }
  // Generate N floats on device between 0.0 and 1.0
  CURAND_CHECK(hiprandGenerateUniform(curand_gen, d_rands, num_samples));
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "RNG per sample runtime [gpu] = " << t.Seconds() << " sec\n";

  // allocate a RNG for each warp
  hiprandState *d_states;
  hipMalloc(&d_states, sizeof(hiprandState)*nblocks*nwarps);
  setup_kernel<<<nblocks, nwarps>>>(d_states);
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  t.Start();
  std::cout << "Launching kernel: num_samples=" << num_samples << "\n";
  k_path_sample<<<nblocks, nthreads>>>(k, d_states, num_samples, d_rands, gg, frontier_list, md, d_counter, d_num_hits);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "runtime [gpu] = " << t.Seconds() << " sec\n";

  CUDA_SAFE_CALL(hipMemcpy(&h_counter, d_counter, sizeof(CounterType), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(&h_num_hits, d_num_hits, sizeof(AccType), hipMemcpyDeviceToHost));

  std::cout << "counter = " << h_counter << " avg_count_per_sample = " << double(h_counter)/double(num_samples) << "\n";
  std::cout << "num_hits = " << h_num_hits << " hit_rate = " << double(h_num_hits)/double(num_samples) << "\n";

  double estimate = h_counter * CounterType(g.E()) / CounterType(num_samples);

  // clean up
  CUDA_SAFE_CALL(hipFree(d_counter));
  CURAND_CHECK(hiprandDestroyGenerator(curand_gen));
  CUDA_SAFE_CALL(hipFree(d_rands));
  return estimate;
}

