#include "hip/hip_runtime.h"
#include "graph.h"
#include "sample.hh"

double sample_house(Graph &g, eidType num_samples);

int main(int argc, char* argv[]) {
  if (argc < 3) {
    std::cout << "Usage: " << argv[0] << " <graph> <num_samples>\n";
    std::cout << "Example: " << argv[0] << " ../../inputs/mico/graph 3 1000\n";
    exit(1);
  }
  Graph g(argv[1]);
  g.print_meta_data();
  int64_t num_samples = atoi(argv[2]);

  Timer t;
  auto m = g.init_edgelist(true);
  std::cout << "Number of different edges: " << m << "\n";

  t.Start();
  uint64_t total = 0;
  auto estimate = sample_house(g, num_samples);
  t.Stop();
  std::cout << "Total runtime [data-copy + gpu-compute] = " << t.Seconds() << " sec\n";
  //std::cout << "Estimated count " << FormatWithCommas(estimate) << "\n";
  std::cout << "Estimated count " << estimate << "\n";
}

#include <hipcub/hipcub.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "graph_gpu.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

#define BLK_SZ BLOCK_SIZE
typedef double CounterType;
typedef hipcub::BlockReduce<CounterType, BLK_SZ> BlockReduce;

template <typename T>
__device__ T random_select_single(T n, hiprandState &state) {
  return T(ceil(hiprand_uniform(&state) * n) - 1);
}

__global__ void setup_kernel(hiprandState *states) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hiprand_init(1234, idx, 0, &states[idx]);
}

__global__ void house_sample(hiprandState* states, eidType n_samples, float* edges_rng, GraphGPU g, vidType *vlists, vidType max_deg, CounterType *total, AccType *num_hits) {
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int thread_id   = blockIdx.x * blockDim.x + threadIdx.x;
  int warp_id     = thread_id   / WARP_SIZE;                // global warp index
  int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
  int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
  int num_warps   = (BLK_SZ / WARP_SIZE) * gridDim.x;       // total number of active warps

  vidType *vlist  = &vlists[int64_t(warp_id)*int64_t(max_deg)*2];
  vidType *vlist1 = &vlists[int64_t(warp_id)*int64_t(max_deg)*2+int64_t(max_deg)];
  CounterType counter = 0;
  __shared__ CounterType scale[WARPS_PER_BLOCK];
  __shared__ vidType idx0[WARPS_PER_BLOCK];
  __shared__ vidType idx1[WARPS_PER_BLOCK];
  //__shared__ vidType idx2[WARPS_PER_BLOCK];
  __shared__ vidType vs02[WARPS_PER_BLOCK][2];
  __shared__ vidType vs12[WARPS_PER_BLOCK][2];
  eidType m = g.E() / 2;
  //if (thread_lane == 0) idx1[warp_lane] = 0; __syncwarp();

  // each warp draws one sample
  for (eidType sample_id = warp_id; sample_id < n_samples; sample_id += num_warps) {
    // sample an edge
    eidType eid = ceil(edges_rng[sample_id]*m) - 1;
    auto v0 = g.get_src(eid);
    auto v1 = g.get_dst(eid);
    vidType v0_size = g.getOutDegree(v0);
    if (v0_size < 3) continue;
    vidType v1_size = g.getOutDegree(v1);
    if (v1_size < 3) continue;

    // intersection of v0 and v1's neighbors
    auto c0 = intersect(g.N(v0), v0_size, g.N(v1), v1_size, vlist);
    __syncwarp();
    if (c0 == 0) continue;

    // sample a common neighbor of v0 and v1's as v2
    if (thread_lane == 0) idx0[warp_lane] = ceil(hiprand_uniform(&states[warp_id]) * c0) - 1;
    __syncwarp();
    vidType v2 = vlist[idx0[warp_lane]];
 
    if (thread_lane == 0) scale[warp_lane] = CounterType(c0); __syncwarp();

    auto u = v1 > v2 ? v2 : v1;
    auto w = v1 > v2 ? v1 : v2;
    if (thread_lane == 0) {
      vs12[warp_lane][0] = u;
      vs12[warp_lane][1] = w;
    }
    __syncwarp();

    auto c1 = v0_size - 2;
    //auto c1 = difference_set(g.N(v0), v0_size, vs12[warp_lane], 2, vlist);
    if (thread_lane == 0) idx1[warp_lane] = ceil(hiprand_uniform(&states[warp_id]) * c1) - 1;
    __syncwarp();
    auto v3 = difference_sample(g.N(v0), v0_size, vs12[warp_lane], 2, idx1[warp_lane]);
    //vidType v3 = vlist[idx1[warp_lane]];
    vidType v3_degree = g.getOutDegree(v3);
 
    if (thread_lane == 0) scale[warp_lane] *= c1; __syncwarp();

    u = v0 > v2 ? v2 : v0;
    w = v0 > v2 ? v0 : v2;
    if (thread_lane == 0) {
      vs02[warp_lane][0] = u;
      vs02[warp_lane][1] = w;
    }
    __syncwarp();
    //auto cx = difference_set(g.N(v1), v1_size, vs02[warp_lane], 2, vlist1);
    auto cx = difference_set(g.N(v1), v1_size, vs02[warp_lane], 2, vlist1);
    __syncwarp();
    //if (cx != v1_size - 2 && thread_lane == 0)
    //  printf("sample_id=%ld, wid=%d, eid=%ld, v0=%d (deg=%d), v1=%d (deg=%d), idx0=%d, v2=%d, idx1=%d, v3=%d, c0=%d, c1=%d, cx=%d\n", 
    //                              sample_id, warp_id, eid, v0, v0_size, v1, v1_size, idx0[warp_lane], v2, idx1[warp_lane], v3, c0, c1, cx);
    //assert(cx == v1_size - 2);
    auto num = intersect_num(g.N(v3), v3_degree, vlist1, cx);
    auto c2 = warp_reduce<vidType>(num);
    //if (c2 < 1) continue;
    //__syncwarp();

    if (thread_lane == 0) atomicAdd(num_hits, 1);

    if (thread_lane == 0) scale[warp_lane] *= c2; __syncwarp();
    if (thread_lane == 0) counter += scale[warp_lane];
    //auto scale = CounterType(c0) * CounterType(c1) * CounterType(c2);
    //if (thread_lane == 0) counter += scale;
    //if (thread_lane == 0)
    //  printf("sample_id=%ld, wid=%d, eid=%ld, v0=%d (deg=%d), v1=%d (deg=%d), idx1=%d, v2=%d, idx3=%d, v3=%d, c0=%d, c1=%d, c2=%d, cx=%d, scale=%f\n", 
    //                              sample_id, warp_id, eid, v0, v0_size, v1, v1_size, idx1[warp_lane], v2, idx3, v3, c0, c1, c2, cx, scale);
    __syncwarp();
  }
  auto block_num = BlockReduce(temp_storage).Sum(counter);
  if (threadIdx.x == 0) atomicAdd(total, block_num);
}

#define PAR_FACTOR 10
double sample_house(Graph &g, eidType num_samples) {
  vidType nv = g.num_vertices();
  eidType ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t memsize = print_device_info(0);
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";
  if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

  Timer t;
  t.Start();
  GraphGPU gg(g);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "time[copy graph to gpu] = " << t.Seconds() << " sec\n";

  t.Start();
  auto m = g.E()/2;
  gg.copy_edgelist_to_device(m, g);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "copy edgelist to gpu: " << m << " edges\n";
  std::cout << "time[copy edgelist to gpu] = " << t.Seconds() << " sec\n";

  size_t nwarps = WARPS_PER_BLOCK;
  size_t nthreads = BLK_SZ;
  size_t nblocks = (num_samples-1)/WARPS_PER_BLOCK+1;
  int max_blocks_per_SM = 0;
  max_blocks_per_SM = maximum_residency(house_sample, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  size_t max_resident_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  std::cout << "max_resident_blocks = " << max_resident_blocks << "\n";
  if (nblocks > max_resident_blocks*PAR_FACTOR) nblocks = max_resident_blocks*PAR_FACTOR;
  if (nblocks > 65536) nblocks = 65536;

  size_t per_block_vlist_size = nwarps * size_t(md) * sizeof(vidType) * 2;
  size_t nb = (memsize - mem_graph) / per_block_vlist_size;
  //if (nb < nblocks) nblocks = nb;
  size_t list_size = nblocks * per_block_vlist_size;
  std::cout << "frontier list size: " << list_size << " Bytes, " << list_size/(1024*1024) << " MB\n";
  vidType *frontier_list; // each warp has a vertex set; each set has size of max_degree
  CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  CounterType h_counter = 0, *d_counter;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_counter, sizeof(CounterType)));
  CUDA_SAFE_CALL(hipMemcpy(d_counter, &h_counter, sizeof(CounterType), hipMemcpyHostToDevice));
  AccType h_num_hits = 0, *d_num_hits;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_num_hits, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_num_hits, &h_num_hits, sizeof(AccType), hipMemcpyHostToDevice));

  // generate random numbers for edge sampling
  t.Start();
  float* d_rands;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_rands, num_samples * sizeof(float)));
  hiprandGenerator_t curand_gen; // used to generate random numbers on GPU
  auto status = hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  if (status != HIPRAND_STATUS_SUCCESS) {
    printf("Error encountered in generating handle\n");  
  }
  // set seed
  status = hiprandSetPseudoRandomGeneratorSeed(curand_gen ,1234ULL);  
  if (status != HIPRAND_STATUS_SUCCESS) {
    printf("Error encountered in setting seed\n");
  }
  // Generate N floats on device between 0.0 and 1.0
  CURAND_CHECK(hiprandGenerateUniform(curand_gen, d_rands, num_samples));
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "RNG per sample runtime [gpu] = " << t.Seconds() << " sec\n";

  // allocate a RNG for each warp
  hiprandState *d_states;
  hipMalloc(&d_states, sizeof(hiprandState)*nblocks*nwarps);
  setup_kernel<<<nblocks, nwarps>>>(d_states);
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  std::cout << "CUDA sampling house (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  t.Start();
  std::cout << "Launching kernel: num_samples=" << num_samples << "\n";
  house_sample<<<nblocks, nthreads>>>(d_states, num_samples, d_rands, gg, frontier_list, md, d_counter, d_num_hits);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  std::cout << "runtime [gpu] = " << t.Seconds() << " sec\n";

  CUDA_SAFE_CALL(hipMemcpy(&h_counter, d_counter, sizeof(CounterType), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(&h_num_hits, d_num_hits, sizeof(AccType), hipMemcpyDeviceToHost));

  std::cout << "counter = " << h_counter << " avg_count_per_sample = " << double(h_counter)/double(num_samples) << "\n";
  std::cout << "num_hits = " << h_num_hits << " hit_rate = " << double(h_num_hits)/double(num_samples) << "\n";

  double estimate = (h_counter * double(m)) / double(num_samples);

  // clean up
  CUDA_SAFE_CALL(hipFree(d_counter));
  CURAND_CHECK(hiprandDestroyGenerator(curand_gen));
  CUDA_SAFE_CALL(hipFree(d_rands));
  return estimate;
}

